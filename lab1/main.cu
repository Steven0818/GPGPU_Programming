#include "hip/hip_runtime.h"
#include <random>
#include <vector>
#include <tuple>
#include <cstdio>
#include <stdlib.h>
#include <cstdlib>
#include <functional>
#include <algorithm>
#include "../utils/SyncedMemory.h"
#include "../utils/Timer.h"
#include "counting.h"
#include <iostream>
#include <fstream>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
using namespace std;

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

template <typename Engine>
tuple<vector<char>, vector<int>, vector<int>> GenerateTestCase(Engine &eng, const int N) {
	poisson_distribution<int> pd(14.0);
	bernoulli_distribution bd(0.1);
	uniform_int_distribution<int> id1(1, 20);
	uniform_int_distribution<int> id2(1, 5);
	uniform_int_distribution<int> id3('a', 'z');
	tuple<vector<char>, vector<int>, vector<int>> ret;
	auto &text = get<0>(ret);
	auto &pos = get<1>(ret);
	auto &head = get<2>(ret);
	auto gen_rand_word_len = [&] () -> int {
		return max(1, min(500, pd(eng) - 5 + (bd(eng) ? id1(eng)*20 : 0)));
	};
	auto gen_rand_space_len = [&] () -> int {
		return id2(eng);
	};
	auto gen_rand_char = [&] () {
		return id3(eng);
	};
	auto AddWord = [&] () {
		head.push_back(text.size());
		int n = gen_rand_word_len();
		for (int i = 0; i < n; ++i) {
			text.push_back(gen_rand_char());
			pos.push_back(i+1);
		}
	};
	auto AddSpace = [&] () {
		int n = gen_rand_space_len();
		for (int i = 0; i < n; ++i) {
			text.push_back('\n');
			pos.push_back(0);
		}
	};

	AddWord();
	while (text.size() < N) {
		AddSpace();
		AddWord();
	}
	return ret;
}



int main(int argc, char **argv)
{
	// Initialize random text
	default_random_engine engine(12345);
	auto text_pos_head = GenerateTestCase(engine, 400000); // 40 MB data
	vector<char> &text = get<0>(text_pos_head);
	vector<int> &pos = get<1>(text_pos_head);
	vector<int> &head = get<2>(text_pos_head);

	// Prepare buffers
	int n = text.size();
	char filename[]="text.txt";
    fstream fp;
    fp.open(filename, ios::out);//開啟檔案
    if(!fp){//如果開啟檔案失敗，fp為0；成功，fp為非0
        cout<<"Fail to open file: "<<filename<<endl;
    }
   
	for (vector<char>::iterator it = text.begin() ; it != text.end(); ++it){
		fp<<*it;//寫入字串
	}
    fp.close();
	char *text_gpu;
	hipMalloc(&text_gpu, sizeof(char)*n);
	SyncedMemory<char> text_sync(text.data(), text_gpu, n);
	text_sync.get_cpu_wo(); // touch the cpu data
	MemoryBuffer<int> pos_yours(n), head_yours(n);
	auto pos_yours_sync = pos_yours.CreateSync(n);
	auto head_yours_sync = head_yours.CreateSync(n);

	// Create timers
	Timer timer_count_position;

	// Part I
	timer_count_position.Start();
	int *pos_yours_gpu = pos_yours_sync.get_gpu_wo();
	CountPosition(text_sync.get_gpu_ro(), pos_yours_gpu, n);
	timer_count_position.Pause();
	CHECK;
	//////     write pos file     /////
	/*const int *pos_yours_cpu = pos_yours_sync.get_cpu_ro();
	char filename2[] = "pos.txt";
	fp.open(filename2, ios::out);
    if(!fp){
        cout<<"Fail to open file: "<<filename2<<endl;
    }
   
	for (int i=0;i<n;i++){
		fp<<pos_yours_cpu[i]<<endl;//寫入字串
	}*/
	///////////////////////////////////
	printf_timer(timer_count_position);
	// Part I check
	const int *golden = pos.data();
	const int *yours = pos_yours_sync.get_cpu_ro();
	int n_match1 = mismatch(golden, golden+n, yours).first - golden;
	if (n_match1 != n) {
		puts("Part I WA!");
		copy_n(golden, n, pos_yours_sync.get_cpu_wo());
	}

	// Part II
	int *head_yours_gpu = head_yours_sync.get_gpu_wo();
	int n_head = ExtractHead(pos_yours_sync.get_gpu_ro(), head_yours_gpu, n);
	CHECK;
	// Part II check
	do {
		if (n_head != head.size()) {
			n_head = head.size();
			puts("Part II WA (wrong number of heads)!");
		} else {
			int n_match2 = mismatch(head.begin(), head.end(), head_yours_sync.get_cpu_ro()).first - head.begin();
			if (n_match2 != n_head) {
				puts("Part II WA (wrong heads)!");
			} else {
				break;
			}
		}
		copy_n(head.begin(), n_head, head_yours_sync.get_cpu_wo());
	} while(false);

	// Part III
	// Do whatever your want

	Part3(text_gpu, pos_yours_sync.get_gpu_rw(), head_yours_sync.get_gpu_rw(), n, n_head);

	thrust::device_ptr<const char> pos_d(text_gpu);
    thrust::copy(pos_d,pos_d+n,std::ostream_iterator<char>(std::cout, ""));
	
	char* text_yours_cpu = (char*)malloc(n*sizeof(char));
	hipMemcpy(text_yours_cpu,text_gpu,sizeof(char)*n, hipMemcpyDeviceToHost);
	cout<< text_yours_cpu[1]<<endl;
	char filename2[] = "newText.txt";

	fp.open(filename2, ios::out);
    if(!fp){
        cout<<"Fail to open file: "<<filename2<<endl;
    }
   
	for (int i =0;i<n;i++){
		cout<<text_yours_cpu[i];//寫入字串
		fp<<"i";
	}

	fp.close();

	const int *pos_yours_cpu = pos_yours_sync.get_cpu_ro();
	char filename3[] = "pos.txt";
	fp.open(filename3, ios::out);
    if(!fp){
        cout<<"Fail to open file: "<<filename3<<endl;
    }
   
	for (int i=0;i<n;i++){
		fp<<pos_yours_cpu[i]<<endl;//寫入字串
	}
	CHECK;

	hipFree(text_gpu);
	return 0;
}
